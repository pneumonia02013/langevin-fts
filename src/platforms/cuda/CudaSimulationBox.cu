/*-------------------------------------------------------------
* This class defines simulation box parameters and provide
* methods that compute inner product in a given geometry.
*--------------------------------------------------------------*/
#include <iostream>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include "CudaSimulationBox.h"
#include "CudaCommon.h"

//----------------- Constructor -----------------------------
CudaSimulationBox::CudaSimulationBox(
    std::vector<int> nx, std::vector<double> lx)
    : SimulationBox(nx, lx)
{
    initialize();
}
void CudaSimulationBox::initialize()
{
    sum = new double[n_grid];
    gpu_error_check(hipMalloc((void**)&d_dv, sizeof(double)*n_grid));
    gpu_error_check(hipMemcpy(d_dv, dv,      sizeof(double)*n_grid,hipMemcpyHostToDevice));

    // temporal storage
    gpu_error_check(hipMalloc((void**)&d_sum, sizeof(double)*n_grid));
    gpu_error_check(hipMalloc((void**)&d_multiple, sizeof(double)*n_grid));
}
//----------------- Destructor -----------------------------
CudaSimulationBox::~CudaSimulationBox()
{
    delete[] sum;
    hipFree(d_dv);
    hipFree(d_sum);
    hipFree(d_multiple);
}
//-----------------------------------------------------------
void CudaSimulationBox::set_lx(std::vector<double> new_lx)
{
    SimulationBox::set_lx(new_lx);
    gpu_error_check(hipMemcpy(d_dv, dv,  sizeof(double)*n_grid,hipMemcpyHostToDevice));
}
//-----------------------------------------------------------
double CudaSimulationBox::integral_gpu(double *d_g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    thrust::device_ptr<double> temp_gpu_ptr(d_sum);
    
    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_dv, 1.0, n_grid);
    return thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + n_grid);
}
//-----------------------------------------------------------
double CudaSimulationBox::inner_product_gpu(double *d_g, double *d_h)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_multiple, d_g, d_h, 1.0, n_grid);
    return CudaSimulationBox::integral_gpu(d_multiple);
}
//-----------------------------------------------------------
double CudaSimulationBox::mutiple_inner_product_gpu(int n_comp, double *d_g, double *d_h)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    mutiple_multi_real<<<N_BLOCKS, N_THREADS>>>(n_comp, d_multiple, d_g, d_h, 1.0, n_grid);
    return CudaSimulationBox::integral_gpu(d_multiple);
}
