#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <string>
#include "CudaCommon.h"

CudaCommon::CudaCommon()
{
    const char *ENV_N_BLOCKS  = getenv("LFTS_GPU_NUM_BLOCKS");
    const char *ENV_N_THREADS = getenv("LFTS_GPU_NUM_THREADS");

    std::string env_var_n_blocks (ENV_N_BLOCKS  ? ENV_N_BLOCKS  : "");
    std::string env_var_n_threads(ENV_N_THREADS ? ENV_N_THREADS : "");

    if (env_var_n_blocks.empty())
        this->n_blocks = 256;
    else
        this->n_blocks = std::stoi(env_var_n_blocks);

    if (env_var_n_threads.empty())
        this->n_threads = 256;
    else
        this->n_threads = std::stoi(env_var_n_threads);
}
void CudaCommon::set(int n_blocks, int n_threads, int process_idx)
{
    int devices_count;
    hipError_t err;

    this->set_n_blocks(n_blocks);
    this->set_n_threads(n_threads);

    // change GPU setting
    err = hipGetDeviceCount(&devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
    err = hipSetDevice(process_idx%devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
}
int CudaCommon::get_n_blocks()
{
    return n_blocks;
}
int CudaCommon::get_n_threads()
{
    return n_threads;
}
void CudaCommon::set_n_blocks(int n_blocks)
{
    this->n_blocks = n_blocks;
}
void CudaCommon::set_n_threads(int n_threads)
{
    this->n_threads = n_threads;
}
void CudaCommon::set_idx(int process_idx)
{
    int devices_count;
    hipError_t err;

    // change GPU setting
    err = hipGetDeviceCount(&devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
    err = hipSetDevice(process_idx%devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
}
__global__ void multi_real(double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void mutiple_multi_real(int n_comp,
                          double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {  
        dst[i] = a * src1[i] * src2[i];
        for(int n = 1; n < n_comp; n++)
            dst[i] += a * src1[i+n*M] * src2[i+n*M];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void divide_real(double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i]/src2[i];
        i += blockDim.x * gridDim.x;
    }
}
__global__ void add_multi_real(double* dst,
                             double* src1,
                             double* src2,
                             double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void lin_comb(double* dst,
                        double a,
                        double* src1,
                        double b,
                        double* src2,
                        const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void add_lin_comb(double* dst,
                           double a,
                           double* src1,
                           double b,
                           double* src2,
                           const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multi_complex_real(ftsComplex* dst,
                                 double* src, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i].x = dst[i].x * src[i];
        dst[i].y = dst[i].y * src[i];
        i += blockDim.x * gridDim.x;
    }
}
__global__ void multi_complex_conjugate(double* dst,
                                 ftsComplex* src1,
                                 ftsComplex* src2, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = src1[i].x * src2[i].x + src1[i].y * src2[i].y;
        i += blockDim.x * gridDim.x;
    }
}
