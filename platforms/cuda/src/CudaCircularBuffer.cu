#include "hip/hip_runtime.h"
#include <algorithm>
#include "CudaCircularBuffer.h"

CudaCircularBuffer::CudaCircularBuffer(int length, int width)
{
    this->length = length;
    this->width = width;
    this->start = 0;
    this->n_items = 0;

    hipMalloc((void**)&elems_d, sizeof(double)*length*width);
}
CudaCircularBuffer::~CudaCircularBuffer()
{
    hipFree(elems_d);
}
void CudaCircularBuffer::reset()
{
    start = 0;
    n_items = 0;
}
void CudaCircularBuffer::insert(double* new_arr)
{
    int i = (start+n_items)%length;
    hipMemcpy(&elems_d[i*width], new_arr, sizeof(double)*width, hipMemcpyHostToDevice);
    if (n_items == length)
        start = (start+1)%length;
    n_items = min(n_items+1, length);
}
double* CudaCircularBuffer::get_array(int n)
{
    int i = (start+n_items-n-1+length)%length;
    return &elems_d[i*width];
}
