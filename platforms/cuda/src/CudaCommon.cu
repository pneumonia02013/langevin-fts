#include "hip/hip_runtime.h"

#include <iostream>
#include "CudaCommon.h"

CudaCommon::CudaCommon()
{
    this->n_blocks = 256;
    this->n_threads = 256;
}
void CudaCommon::set(int n_blocks, int n_threads, int process_idx)
{
    int devices_count;
    hipError_t err;

    CudaCommon &pp = CudaCommon::get_instance();
    pp.set_n_blocks(n_blocks);
    pp.set_n_threads(n_threads);

    // change GPU setting
    err = hipGetDeviceCount(&devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
    err = hipSetDevice(process_idx%devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
}
int CudaCommon::get_n_blocks()
{
    return n_blocks;
}
int CudaCommon::get_n_threads()
{
    return n_threads;
}
void CudaCommon::set_n_blocks(int n_blocks)
{
    this->n_blocks = n_blocks;
}
void CudaCommon::set_n_threads(int n_threads)
{
    this->n_threads = n_threads;
}
void CudaCommon::set_idx(int process_idx)
{
    int devices_count;
    hipError_t err;

    // change GPU setting
    err = hipGetDeviceCount(&devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
    err = hipSetDevice(process_idx%devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
}

void CudaCommon::display_info()
{
    int device;
    int devices_count;
    struct hipDeviceProp_t prop;
    hipError_t err;

    CudaCommon &pp = CudaCommon::get_instance();
    const int N_BLOCKS = pp.get_n_blocks();
    const int N_THREADS = pp.get_n_threads();

    // get GPU info
    err = hipGetDeviceCount(&devices_count);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
    err = hipGetDevice(&device);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess)
    {
        std::cout<< hipGetErrorString(err) << std::endl;
        exit (1);
    }

    std::cout<< "---------- CUDA Setting and Device Information ----------" << std::endl;
    std::cout<< "N_BLOCKS, N_THREADS: " << N_BLOCKS << ", " << N_THREADS << std::endl;

    std::cout<< "DeviceCount: " << devices_count << std::endl;
    printf( "Device %d : \t\t\t\t%s\n", device, prop.name );
    std::cout<< "Compute capability version : \t\t" << prop.major << "." << prop.minor << std::endl;
    std::cout<< "Multiprocessor : \t\t\t" << prop.multiProcessorCount << std::endl;

    std::cout<< "Global memory : \t\t\t" << prop.totalGlobalMem/(1024*1024) << " MBytes" << std::endl;
    std::cout<< "Constant memory : \t\t\t" << prop.totalConstMem << " Bytes" << std::endl;
    std::cout<< "Shared memory per block : \t\t" << prop.sharedMemPerBlock << " Bytes" << std::endl;
    std::cout<< "Registers available per block : \t" << prop.regsPerBlock << std::endl;

    std::cout<< "Warp size : \t\t\t\t" << prop.warpSize << std::endl;
    std::cout<< "Maximum threads per block : \t\t" << prop.maxThreadsPerBlock << std::endl;
    std::cout<< "Max size of a thread block (x,y,z) : \t(";
    std::cout<< prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")\n";
    std::cout<< "Max size of a grid size    (x,y,z) : \t(";
    std::cout<< prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")\n";

    //if(prop.deviceOverlap)
    //{
    //std::cout<< "Device overlap : \t\t\t Yes" << std::endl;
    //}
    //else
    //{
    //std::cout<< "Device overlap : \t\t\t No" << std::endl;
    //}

    if (N_THREADS > prop.maxThreadsPerBlock)
    {
        std::cout<< "'threads_per_block' cannot be greater than 'Maximum threads per block'" << std::endl;
        exit (1);
    }

    if (N_BLOCKS > prop.maxGridSize[0])
    {
        std::cout<< "The number of blocks cannot be greater than 'Max size of a grid size (x)'" << std::endl;
        exit (1);
    }
    if (prop.warpSize < 32)
    {
        std::cout<< "'Warp size' cannot be less than 32 due to synchronization in 'multi_inner_product_kernel'." << std::endl;
        exit (1);
    }

    if (N_THREADS > 1024)
    {
        std::cout<<"'threads_per_block' cannot be greater than 1024 because of 'multi_inner_product_kernel'." << std::endl;
        exit (1);
    }
}
__global__ void multiReal(double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void mutipleMultiReal(int n_comp,
                          double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {  
        dst[i] = a * src1[i] * src2[i];
        for(int n = 1; n < n_comp; n++)
            dst[i] += a * src1[i+n*M] * src2[i+n*M];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void divideReal(double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i]/src2[i];
        i += blockDim.x * gridDim.x;
    }
}
__global__ void addMultiReal(double* dst,
                             double* src1,
                             double* src2,
                             double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void linComb(double* dst,
                        double a,
                        double* src1,
                        double b,
                        double* src2,
                        const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void addLinComb(double* dst,
                           double a,
                           double* src1,
                           double b,
                           double* src2,
                           const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multiComplexReal(ftsComplex* a,
                                 double* b, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        a[i].x = a[i].x * b[i];
        a[i].y = a[i].y * b[i];
        i += blockDim.x * gridDim.x;
    }
}
